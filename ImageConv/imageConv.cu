#include "hip/hip_runtime.h"
// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// Includes CUDA
#include <hip/hip_runtime.h>
// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

void cpu_image_convolution(float *hData, float *hOutputData, int kernalDim, float *kernal, int height, int width, float factor){
    int kernalX = kernalDim/2;
    int kernalY = kernalDim/2;
    float sum = 0;
    int kernalval = 0;
    int ii = 0;
    int jj = 0;
    for(int i = 0; i < height; i++){
        for(int j = 0; j < width; j++){
            kernalval = 0;
            sum = 0;
            for(int x = -kernalX; x <= kernalX; ++x){
                for(int y = -kernalY; y <= kernalY; ++y){
                    ii = i + x;
                    jj = j + y;
                    if((ii >= 0 && ii < height) && (jj >= 0 && jj < width)){
                        sum += (hData[ii*height + jj] * (factor * kernal[kernalval]));
                    }                    
                    kernalval++;
                }   
            }
            hOutputData[i*height + j] = sum;
        }
    }
}

__global__ void gpu_image_convolution(float *dData, float *dOutputData, int kernalDim, float *kernal, int height, int width, float factor){
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int kernalX = kernalDim/2;
    int kernalY = kernalDim/2;
    float sum = 0;
    int kernalval = 0;
    int ii = 0;
    int jj = 0;
    if(i < height && j < width){
        for(int x = -kernalX; x <= kernalX; ++x){
            for(int y = -kernalY; y <= kernalY; ++y){
                ii = i + x;
                jj = j + y;
                if((ii >= 0 && ii < height) && (jj >= 0 && jj < width)){
                    sum += (dData[ii*height + jj] * (factor * kernal[kernalval]));
                }                    
                kernalval++;
            }   
        }
        dOutputData[i*height + j] = sum;
    }
}

__constant__ float const_kernal[1024];
__global__ void gpu_image_convolution_shared(float *dData, float *dOutputData, int kernalDim, int height, int width, float factor){
	__shared__ float sharedMemory[16][16];
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int kernalX = kernalDim/2;
    int kernalY = kernalDim/2;
    float sum = 0;
    int kernalval = 0;
    int ii = 0;
    int jj = 0;
    if(i < height && j < width){       
        sharedMemory[threadIdx.x][threadIdx.y] = dData[i*height + j];
    	__syncthreads();

        for(int x = -kernalX; x <= kernalX; ++x){
            for(int y = -kernalY; y <= kernalY; ++y){
                ii = threadIdx.x + x;
                jj = threadIdx.y + y;
                if((ii >= 0 && ii < 16) && (jj >= 0 && jj < 16)){
                    sum += (sharedMemory[ii][jj] * (factor * const_kernal[kernalval]));
                }else{
                    int iii = ii - threadIdx.x + i;
                    int jjj = jj - threadIdx.y + j;
                    if(iii >= 0 && iii < height && jjj >=0 && jjj < width){
                        sum += (dData[iii*height + jjj] * (factor * const_kernal[kernalval]));
                    }
                }                    
                kernalval++;
            }   
        }
        dOutputData[i*height + j] = sum;
    }
}

int main(int argc, char **argv){
    int kernalChoice;
    char inputFile[1000];
    char outputFile[1000];
    float factor = 1;
    // program kernalChoice, filename.pgm, outputname.pgm, factor
    if(argc != 5){
        printf("Usage error: imgConv kernalChoice inputfile.pgm outputfile.pgm factor");
        exit(EXIT_FAILURE);
    }else{
        kernalChoice = atoi(argv[1]);
        strcpy(inputFile, argv[2]);
        strcpy(outputFile, argv[3]);
        factor = atof(argv[4]);
    }
    
    const char *imageName = inputFile;
    float *hData = NULL;
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageName, "/data");

    if(imagePath == NULL){
        printf("Unable to find image: %s\n", imageName);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(imagePath, &hData, &width, &height);
    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n", imageName, width, height);

    float sharpen[] = {0.0,-1.0,0.0,
                         -1.0,5.0,-1.0,
                          0.0,-1.0,0.0}; // Sharpen

    float emboss_three[] = {-2.0,-1.0,0.0,
                     -1.0,1.0,1.0,
                      0.0,1.0,2.0}; // Emboss 3x3AS

    float emboss_five[] = {1,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,-1,0,0,0,0,0,-1}; // Emboss
    float average_five[] = {1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1}; // Average

    float *kernal[4] = {sharpen, emboss_three, emboss_five, average_five};

    int kernalDim = 0;
    if(kernalChoice == 0 || kernalChoice == 1){
        kernalDim = 3;
    }else if(kernalChoice == 2 || kernalChoice == 3){
        kernalDim = 5;
    }else{
        printf("Invalid kernal choice!\n");
        exit(EXIT_FAILURE);
    }

    float *hOutputCPU = (float *) malloc(sizeof(float) * size);
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);    
    cpu_image_convolution(hData, hOutputCPU, kernalDim, kernal[kernalChoice],height, width, factor);
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms) \n", sdkGetTimerValue(&timer));
    printf("%.2f Mpixels/sec\n",
           (width *height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
    float timeSerial = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);

    char outputFilename[1024];
    const char *outPath = "./output/cpu_";
    strcpy(outputFilename, outPath);
    strcpy(outputFilename + strlen(outPath), outputFile);
    sdkSavePGM(outputFilename, hOutputCPU, width, height);
    printf("Wrote '%s'\n", outputFilename);

    dim3 blockSize(16, 16);
    dim3 gridSize((height + blockSize.y - 1) / blockSize.y, (width + blockSize.x - 1) / blockSize.x);
    float *dData = NULL;
    float *dOutputData = NULL;
    float *dKernal = NULL;
    float *hOutputData = (float *) malloc(size);
    checkCudaErrors(hipMalloc((void **) &dData, size));
    checkCudaErrors(hipMalloc((void **) &dOutputData, size));
    checkCudaErrors(hipMalloc((void **) &dKernal, (kernalDim*kernalDim*sizeof(float))));

    /**
     * Global Memory GPU
     * 
     */
    checkCudaErrors(hipMemcpy(dData, hData, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dKernal, kernal[kernalChoice], (kernalDim*kernalDim * sizeof(float)), hipMemcpyHostToDevice));
    
    hipEvent_t launch_begin, launch_end;
    hipEventCreate(&launch_begin);
    hipEventCreate(&launch_end);

    hipEventRecord(launch_begin,0);
    gpu_image_convolution<<<gridSize, blockSize>>>(dData, dOutputData, kernalDim, dKernal, height, width, factor);
    hipEventRecord(launch_end,0);
    hipDeviceSynchronize();

    float time = 0;
    hipEventElapsedTime(&time, launch_begin, launch_end);
    checkCudaErrors(hipMemcpy(hOutputData, dOutputData, size, hipMemcpyDeviceToHost));
    printf("GPU Time: %f ms\n", time);
    printf("Speedup: %f\n", timeSerial/time);
    printf("%.2f Mpixels/sec\n",
           (width *height / (time / 1000.0f)) / 1e6);
    outPath = "./output/gpu_global_";
    strcpy(outputFilename, outPath);
    strcpy(outputFilename + strlen(outPath), outputFile);
    sdkSavePGM(outputFilename, hOutputData, width, height);
    printf("Wrote '%s'\n", outputFilename);


    /**
     * Shared and constant memory GPU
     * 
     */
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(const_kernal), kernal[kernalChoice],(kernalDim*kernalDim * sizeof(float))));

    hipEventRecord(launch_begin,0);
    gpu_image_convolution_shared<<<gridSize, blockSize>>>(dData, dOutputData, kernalDim, height, width, factor);
    hipEventRecord(launch_end,0);
    hipDeviceSynchronize();

    time = 0;
    hipEventElapsedTime(&time, launch_begin, launch_end);
    checkCudaErrors(hipMemcpy(hOutputData, dOutputData, size, hipMemcpyDeviceToHost));
    printf("GPU Time: %f ms\n", time);
    printf("Speedup: %f\n", timeSerial/time);
    printf("%.2f Mpixels/sec\n",
           (width *height / (time / 1000.0f)) / 1e6);
    outPath = "./output/gpu_shared_";
    strcpy(outputFilename, outPath);
    strcpy(outputFilename + strlen(outPath), outputFile);
    sdkSavePGM(outputFilename, hOutputData, width, height);
    printf("Wrote '%s'\n", outputFilename);

    free(hOutputCPU);
    checkCudaErrors(hipFree((void *) dData));
    checkCudaErrors(hipFree((void *) dOutputData));
    checkCudaErrors(hipFree((void *) dKernal));
    free(hOutputData);

    return 0;
}